//generamos numeros aleatorios

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

// compilar:
// nvcc -o program program.cu -l curand
// Quiero que cada bloque tire blockSize numeros y caluclar pi. 
// Despues promediar con todos los bloques. 
// Finalmente usar los bloques sin promediar.

__global__ void montecarlo_pi(int N, int *count, float *x, float *y)
{
	int index  = blockIdx.x*blockDim.x + threadIdx.x;

	if (index < N){
		if (x[index]*x[index] + y[index]*y[index] <= 1) {
			count[index]=1; //mala practica, mejor inicializar antes y count[]++;
		}
		else count[index]=0;
	}
}

int main(){

	int N = 1<<24; //no deberia poder ir 1<<31 -1 ??
	int blockSize = 512; //cada bloque procesa 6 mb 
	int numBlock = (N + blockSize - 1)/blockSize;

	printf("Bloques %i, Threads %i\n", numBlock, blockSize);
	printf("N %i \n", N);

	//device data
	float *d_x, *d_y;
	//shared memory
	int *count;

	//Alocamos data
	hipMalloc(&d_x, N*sizeof(float));        //4*N bytes
	hipMalloc(&d_y, N*sizeof(float));        //4*N bytes
	hipMallocManaged(&count, N*sizeof(int)); //2*N bytes

	//Defino el generador. 
	hiprandGenerator_t gen;
	
	//Create generetor mersenne twister engine MTGP32
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

	//Set seed
	hiprandSetPseudoRandomGeneratorSeed(gen, 1245ULL); 

	int n_sim = 10;
	float pi_total = 0;
	for (int simulacion=0; simulacion<n_sim; simulacion++){
		
		//Generate N floats on device
		hiprandGenerateUniform(gen, d_x, N);
		hiprandGenerateUniform(gen, d_y, N);
		hipDeviceSynchronize(); //hace falta?
		//Call kernel
		montecarlo_pi<<<numBlock,blockSize>>>(N,count,d_x,d_y);
		hipDeviceSynchronize(); //hace falta?
		float pi_calc=0;
		for (int i=0; i<N; i++){
			if (count[i] !=0 ) pi_calc++;
		}
		printf("pi %f \n", 4.0f*pi_calc/(float)N);
		pi_total+=4.0f*pi_calc/(float)N;
	}
	printf("pi total %f \n", pi_total/(float)n_sim);




	//cleanup
	hiprandDestroyGenerator(gen);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(count);

	return 0;
}