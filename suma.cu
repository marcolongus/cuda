
#include <hip/hip_runtime.h>
#include <stdio.h>

//indicates a function that runs on the divice
// and also is colled from host code (du global)


//ad a single integer
__global__ void add(int *a, int *b, int *c){
	*c = *a + *b; //ver si funciona sin lo sastericos
	int d = *c + 1; 
	printf("kernel d=c+1: %d\n", d);
	printf("kernel c    : %d\n", *c);
}

int main(void){

	int *a, *b, *c;          //host copies of a,b,c
	int size = sizeof(int);

	//allocate space for device copies of a,b,c
	hipMallocManaged(&a, size); //cuando va??? (void **), como ir a cuda Managed, agregar check error. 
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	*a = 1;
	*b = 1;
	*c = 5;
	
	printf("antes del kernel c: %d\n", *c);
	//launch add() kernel on GPU
	add<<<1,1>>>(a,b,c);
	hipDeviceSynchronize();

	printf("despues del kernel c: %d\n", *c);
	//cleanup
	hipFree(a); hipFree(b); hipFree(c);

	return 0;
}


