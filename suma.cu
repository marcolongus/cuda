
#include <hip/hip_runtime.h>
#include <stdio.h>

//indicates a function that runs on the divice
// and also is colled from host code (du global)


//ad a single integer
__global__ void add(int *a, int *b, int *c){
	*c = *a + *b; //ver si funciona sin lo sastericos
	int d = *c + 1; 
	printf("kernel %d\n", d);
}

int main(void){

	int a, b, c;          //host copies of a,b,c
	int *d_a, *d_b, *d_c; //device copies of a, b,c
	int size = sizeof(int);

	//allocate space for device copies of a,b,c
	hipMalloc((void  **)&d_a, size); //cuando va??? (void **), como ir a cuda Managed, agregar check error. 
	hipMalloc((void  **)&d_b, size);
	hipMalloc((void  **)&d_c, size);

	a = 1;
	b = 1;

	//copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	//launch add() kernel on GPU
	add<<<1,1>>>(d_a,d_b,d_c);

	//copy result back to host
	hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

	printf("%i\n", c);
	//cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}


