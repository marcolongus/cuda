#include "hip/hip_runtime.h"
#include <iostream>
#include <time>
#include <random>
#include <math.h>
#include <hiprand.h>

__global__ void setup_kernel(hiprandState *state){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(123456789, index, 0, &state[index]);
}

__global__ void random_kerndel(hiprandState *state, int * count){
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	float x = hiprand_uniform(&state[index]);
	float y = hiprand_uniform(&state[index])
	float r = x*x + y*y;

	if (r <= 1) count++;
}