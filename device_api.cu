#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/// ver por que no anda la mierda esta

__global__ void setup_kernel(curanState *state)
{
	int id =  blockIdx.x*blockDim.x threadIdx.x;
	hiprand_init(1234, id, 0, &state[id]); //same seed.  
}

__global__ void generate_kernel(curanState *state, int *result)
{
	int id =  blockIdx.x*blockDim.x threadIdx.x; 
	int count =0;

	unsigned int x;
	//copy state to local memory
	hiprandState localState = state[id];

	for(int n=0; n<10000; n++){
		x = hiprand(&localState);
		if (x&1) count++;
	}
	state[id] = localState;
	result[id] += count;
}

int main(void){
	int N=64*64;
	int *devResults, *hostResults;

	hiprandState *devStates;

	hostResults = (int*)malloc(&hostResults, N*sizeof(int));
	hipMalloc(&devResults, N*sizeof(int));
	hipMalloc(&devStates , N*sizeof(int));

	hipMemset(devResults , 0, N*sizeof(int));

	setup_kernel<<<64,64>>>(devStates);

	for(int i=0; i<N; i++){
		generate_kernel<<<64,64>>>(devStates, devResults);
	}

	hipMemcpy(hostResults, devResults, N*sizeof(int), hipMemcpyDeviceToHost);

	int total = 0;
	for (int i = 0; i < N; ++i){
		total+=hostResults[i];
	}

	printf("%10.13f\n", (float)total/(float(N)*100000.0f));

	//cleanup
	hipFree(devStates); hipFree(devResults); free(hostResults);

	return 0;
}
