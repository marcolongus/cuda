#include "hip/hip_runtime.h"
//single-precision A*X plus Y: saxoy
#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("bIdx.x %d, bkDim.x %d, thrIdx %d\n, i %d ", blockIdx.x, blockDim.x, threadIdx.x, i);
	if (i<n) { 
		y[i] = a*x[i] + y[i]; 
		//printf("inner %d\n",i);
		printf("bIdx.x %d, thrIdx %d\n, i %d. ", blockIdx.x, threadIdx.x, i);
	}
}

int main(void){

	int N = 1<<3;
	int blockSize = 2;
	int numBlocks = (N+blockSize-1)/blockSize;//at least as many threads as N
	//host 
	float *x,*y;
	//device 
	float *d_x,*d_y;

	//Host memory allocation
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	//Device memory allocation
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));

	for (int i=0; i<N; i++){
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	//copy host to device memory
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	printf("blocks %d, threads %d.\n\n",numBlocks, blockSize);
	//Llamamos kernells en bloques de 1M
	saxpy<<<numBlocks, blockSize>>> (N, 2.0f, d_x, d_y);

	hipMemcpy(y, d_y, N*sizeof(float),hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i=0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 4.0f));
	printf("\n max error %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

	return 0;
}