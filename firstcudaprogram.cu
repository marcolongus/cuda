#include "hip/hip_runtime.h"
//single-precision A*X plus Y: saxoy
#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("bIdx.x %d, bkDim.x %d, thrIdx %d\n, i %d ", blockIdx.x, blockDim.x, threadIdx.x, i);
	if (i<n) { 
		y[i] = a*x[i] + y[i]; 
		//printf("inner %d\n",i);
		printf("bIdx.x %d, thrIdx %d\n, i %d. ", blockIdx.x, threadIdx.x, i);
	}
}

int main(void){
	int N = 1<<8;
	//host 
	float *x,*y;
	//device 
	float *d_x,*d_y;

	//Host memory allocation
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	//Device memory allocation
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));

	for (int i=0; i<N; i++){
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	//copy host to device memory
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	printf("blocks %d, threads %d.\n\n",(N+255)/256,256 );
	//Llamamos kernells en bloques de 1M
	saxpy<<<(N+255)/256, 256>>> (N, 2.0f, d_x, d_y);

	hipMemcpy(y, d_y, N*sizeof(float),hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i=0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 4.0f));
	printf("max error %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

	return 0;
}